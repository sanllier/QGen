#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cuda_error_handler.h"
#include "defs_gpu.h"

//------------------------------------------------------------

__global__ void setInitialKernel( GPUQbit* data, long long size )
{
    long long index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( index >= size )
        return;

    printf( "%d\r\n", size );

    GPUQbit* targetElement = data + index;

    BASETYPE randVal = 0.0; // CRAP
    targetElement->aReal = randVal;
    targetElement->aImag = BASETYPE(0);
    targetElement->bReal = sqrt( BASETYPE(1) - randVal );
    targetElement->bImag = BASETYPE(0);
}

void launchSetInitialKernel( GPUQbit* data, long long size )
{
    dim3 block = dim3 ( CUDA_BLOCK_SIZE );
    dim3 grid = dim3 ( size / CUDA_BLOCK_SIZE + 1 );
    SAFE_KERNEL_CALL( ( setInitialKernel<<< grid, block >>>( ( GPUQbit* )data, size ) ) );
}

//------------------------------------------------------------
